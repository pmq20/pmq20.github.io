#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cstring>
#include <ctime>

#define WARP_SIZE 32

clock_t start,end;

__global__ void
_cuda_parallel_sum(int *in, int num_elements, int *sum)
{
    __syncthreads();
    __shared__ int buffer[WARP_SIZE];
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int lane = threadIdx.x % WARP_SIZE;
    int temp;
    while(globalIdx < num_elements)
    {
        temp = in[globalIdx];
        for (int delta = WARP_SIZE/2; delta > 0; delta /= 2)
        {
             temp+= __shfl_xor(temp, delta);
        }
        if (lane == 0)
        {
            buffer[threadIdx.x / WARP_SIZE] = temp;
        }
        __syncthreads();
        if(threadIdx.x < WARP_SIZE) 
        {
            temp = buffer[threadIdx.x];
            for(int delta = WARP_SIZE / 2; delta > 0; delta /= 2)
            {  
                temp += __shfl_xor(temp, delta);
            }
        }
        if(threadIdx.x == 0)
        {
            atomicAdd(sum, temp);
        }
        globalIdx += blockDim.x * gridDim.x;
        __syncthreads();
    }
}

int cuda_parallel_sum(int * a, int N) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    int num_SMs = prop.multiProcessorCount;
    start = std::clock();
    int batch_size = num_SMs * 1024;
    int padding = (batch_size - (N % batch_size)) % batch_size;
    int * b = new int[N + padding];
    memcpy(b, a, N * sizeof(int));
    memset(b + N, 0, padding * sizeof(int));
    int *d_b;
    hipMalloc( (void**) &d_b, (N + padding) * sizeof(int) );
    hipMemcpy( d_b, b, (N + padding) * sizeof(int), hipMemcpyHostToDevice );
    int result = 0.0;
    int * d_result;
    hipMalloc( (void**) &d_result, sizeof(int) );
    hipMemcpy( d_result, &result, sizeof(int), hipMemcpyHostToDevice );
    _cuda_parallel_sum<<< num_SMs, 1024 >>>(d_b, N + padding, d_result);
    hipMemcpy( &result, d_result, sizeof(int), hipMemcpyDeviceToHost );
    end = std::clock();
    hipFree(d_result);
    hipFree(d_b);
    free(b);
    return result;
}


int cpu_sum(int * a, int N) {
	int sum = 0;
	start = std::clock();
	for(int i = 0; i < N; ++i) {
    sum += a[i];
  }
  end = std::clock();
  return sum;
}

int main() {
  const int N = 10000;
  hipSetDevice(0);
  int * a = new int[N];
  for(int i = 0; i <= N; i++) {
    a[i] = i;
  }
  int result = cuda_parallel_sum(a, N);
  printf("GPU sum: %d - total time: %lfms\n", result, 1000.0 * (end-start) / CLOCKS_PER_SEC);

  for(int i = 0; i <= N; ++i) {
    a[i] = i;
  }
  result = cpu_sum(a, N);
  printf("CPU sum: %d - total time: %lfms\n", result, 1000.0 * (end-start) / CLOCKS_PER_SEC);
  return 0;
}
